#include "hip/hip_runtime.h"
/*extern "C" {} *///it will instruct the compiler to expect C linkage for your C functions, not C++ linkage.
//#include <thrust/find.h>
//#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/execution_policy.h>
//#include <thrust/device_free.h>
#include <stdio.h>
//#include "all_structures.h"
#include "all_structure_cuda.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>


#include<vector>
#include <chrono> 


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;

__device__ int changeFlag1;
__device__ int changeFlag2;


__global__ void initialize(int nodes, int src, RT_Vertex* SSSP, double dummy_dist, int degree_frame)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int d = degree_frame;
	int gridstride = gridDim.x * blockDim.x;

	for (int index = idx; index < nodes; index += gridstride)
	{
		if (index == src) {
			SSSP[index].Root = src;
			SSSP[index].Parent = src;
			SSSP[index].Dist = 0.0;
			SSSP[index].EDGwt = 0;
		}
		else
		{
			SSSP[index].Root = src;
			int temp = index - 1;
			SSSP[index].Parent = (temp) / d;//we get the floor value as Root stores int
			/*//option1: all edge weight distinct and greater than (max edge_wt) * (n-1)
			SSSP[index].EDGwt = dummy_dist + index;
			int temp_dist = dummy_dist + index;
			while (temp > 0)
			{
				temp = temp / d;
				if (temp > 0)
				{
					temp_dist = temp_dist + dummy_dist + temp;
				}

			}
			SSSP[index].Dist = temp_dist;
			d_UpdatedDist[index] = temp_dist;*/

			//option 2: //taking all edges equal and greater than (max edge_wt) * (n-1)
			SSSP[index].EDGwt = dummy_dist;
			double temp_dist = dummy_dist;
			while (temp > 0)
			{
				temp = temp / d;
				if (temp > 0)
				{
					temp_dist = temp_dist + dummy_dist;
				}

			}
			SSSP[index].Dist = temp_dist;

		}

	}
}



__global__ void updateParent(int numS, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int* changeFlag1, int* changeFlag2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int gridstride = gridDim.x * blockDim.x;

	for (int index = idx; index < numS; index += gridstride)
	{

		//get the edge
		int node_1 = allChange_cuda[index].node1;
		int node_2 = allChange_cuda[index].node2;
		double edgeWeight = allChange_cuda[index].edge_wt;
		//reset it to 0
		//int Edgedone = 0;

		///method 1
		//if (SSSP[node_2].Dist > SSSP[node_1].Dist + edgeWeight)
		//{
		//	//Edgedone = 1;
		//	if (SSSP[node_2].Parent != node_1) {
		//		//Update Parent and EdgeWt
		//		SSSP[node_2].Parent = node_1;
		//		SSSP[node_2].EDGwt = edgeWeight;
		//		SSSP[node_2].Dist = SSSP[node_1].Dist + edgeWeight;
		//		*changeFlag1 = 1; //every time node dist is updated, the flag becomes 1
		//		//printf("*changeFlag1: %d", *changeFlag1);
		//		*changeFlag2 = 1;
		//	}
		//}

		///method 1A
		if (SSSP[node_2].Dist > SSSP[node_1].Dist + edgeWeight)
		{
			//Edgedone = 1;
			
				//Update Parent and EdgeWt
				SSSP[node_2].Parent = node_1;
				SSSP[node_2].EDGwt = edgeWeight;
				SSSP[node_2].Dist = SSSP[node_1].Dist + edgeWeight;
				*changeFlag1 = 1; //every time node dist is updated, the flag becomes 1
				//printf("*changeFlag1: %d", *changeFlag1);
				*changeFlag2 = 1;
			
		}

		//method 2
		//if (SSSP[SSSP[node_2].Parent].Dist + SSSP[node_2].EDGwt> SSSP[node_1].Dist + edgeWeight)
		//{
		//	if (SSSP[node_2].Parent != node_1) {
		//				//Update Parent and EdgeWt
		//				SSSP[node_2].Parent = node_1;
		//				SSSP[node_2].EDGwt = edgeWeight;
		//				//SSSP[node_2].Dist = SSSP[node_1].Dist + edgeWeight;
		//				*changeFlag1 = 1; //every time node dist is updated, the flag becomes 1
		//				//printf("*changeFlag1: %d", *changeFlag1);
		//				*changeFlag2 = 1;
		//			}
		//}
	}
}



__global__ void checkDist(int nodes, RT_Vertex* SSSP, int* changeFlag1, int* changeFlag2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int gridstride = gridDim.x * blockDim.x;

	for (int index = idx; index < nodes; index += gridstride)
	{
		int parent = SSSP[index].Parent;
		if (SSSP[index].Dist != SSSP[parent].Dist + SSSP[index].EDGwt)
		{
			SSSP[index].Dist = SSSP[parent].Dist + SSSP[index].EDGwt;
			*changeFlag2 = 1;
			*changeFlag1 = 1;
		//printf("index: %d, dist: %f , parent's dist: %f, edgeweight: %f\n", index, SSSP[index].Dist, SSSP[parent].Dist, SSSP[index].EDGwt);
		}
	}
}


__global__ void helperFunction(int total_block, int thread_per_block, int nodes, int src, RT_Vertex* SSSP, double dummy_dist, int degree_frame, int totalChange, xEdge_cuda* allChange_cuda)
{
	if (threadIdx.x == 0)
	{
		initialize << <total_block, thread_per_block >> > (nodes, src, SSSP, dummy_dist, degree_frame);
		changeFlag1 = 1;
		changeFlag2 = 1;
		int itr = 0, itr2 = 0;

		///Method 1
		for (int i = 0; i < degree_frame; i++)
		{
			updateParent << < total_block, thread_per_block >> > (totalChange, allChange_cuda, SSSP, &changeFlag1, &changeFlag2);
			//hipDeviceSynchronize();
			//printf("after itr %d changeFlag1: %d", itr, changeFlag1);
			itr++;
		}
		hipDeviceSynchronize();
		changeFlag1 = 1;
		changeFlag2 = 1;
		
		while (changeFlag2 == 1)
		{
			changeFlag2 = 0;
			checkDist << <total_block, thread_per_block >> > (nodes, SSSP, &changeFlag1, &changeFlag2);
			hipDeviceSynchronize();
			itr2++;
		}
		//hipDeviceSynchronize();

		while (changeFlag1 == 1)
		{
			changeFlag1 = 0;
			updateParent << < total_block, thread_per_block >> > (totalChange, allChange_cuda, SSSP, &changeFlag1, &changeFlag2);
			hipDeviceSynchronize();
			itr++;
			if (changeFlag1 == 1 /**&& itr % *degree_frame == 0**/)
			{
				changeFlag2 = 1;
				//printf("itr value is: %d \n", itr);
			}
			while (changeFlag2 == 1)
			{
				changeFlag2 = 0;
				checkDist << <total_block, thread_per_block >> > (nodes, SSSP, &changeFlag1, &changeFlag2);
				//hipDeviceSynchronize();
				itr2++;

			}
			hipDeviceSynchronize();
		}

		///Method 1A
		//hipStream_t stream1;
		//hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
		//hipStream_t stream2;
		//hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);
		//changeFlag1 = 1;
		//changeFlag2 = 1;
		//while (changeFlag1 == 1 || changeFlag2 == 1)
		//{
		//	changeFlag1 = 0;
		//	updateParent << < total_block, thread_per_block , 0 , stream1 >> > (totalChange, allChange_cuda, SSSP, &changeFlag1, &changeFlag2);
		//	//hipDeviceSynchronize();
		//	//printf("after itr %d changeFlag1: %d", itr, changeFlag1);
		//	itr++;
		//	changeFlag2 = 0;
		//	checkDist << <total_block, thread_per_block, 0, stream2 >> > (nodes, SSSP, &changeFlag1, &changeFlag2);
		//	//hipDeviceSynchronize();
		//	itr2++;
		//	hipDeviceSynchronize();
		//	
		//}
		//
		//hipStreamDestroy(stream1);
		//hipStreamDestroy(stream2);





		///Method 2
		//for (int i = 0; i < degree_frame; i++)
		//{
		//	updateParent << < total_block, thread_per_block >> > (totalChange, allChange_cuda, SSSP, &changeFlag1, &changeFlag2);
		//	//hipDeviceSynchronize();
		//	//printf("after itr %d changeFlag1: %d", itr, changeFlag1);
		//	itr++;
		//}
		//hipDeviceSynchronize();
		//changeFlag1 = 1;
		//changeFlag2 = 1;
		//while (changeFlag1 == 1)
		//{
		//	while (changeFlag1 == 1)
		//	{
		//		changeFlag1 = 0;
		//		updateParent << < total_block, thread_per_block >> > (totalChange, allChange_cuda, SSSP, &changeFlag1, &changeFlag2);
		//		//hipDeviceSynchronize();
		//		//printf("after itr %d changeFlag1: %d", itr, changeFlag1);
		//		itr++;
		//	}
		//	while (changeFlag2 == 1)
		//	{
		//		changeFlag2 = 0;
		//		checkDist << <total_block, thread_per_block >> > (nodes, SSSP, &changeFlag1, &changeFlag2);
		//		//hipDeviceSynchronize();
		//		itr2++;
		//		//printf("itr2 value: %d changeflag2: %d", itr2, changeFlag2);
		//	}
		//	hipDeviceSynchronize();
		//}
		printf("itr %d itr2: %d", itr, itr2);

	}
}

void edge_update(int* totalChange, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int total_block, int thread_per_block, int* nodes, int* degree_frame);

/*
1st arg: original graph file name
2nd arg: no. of nodes
3rd arg: no. of edges
4th arg: max edge weight
5th arg: degree of frame
6th: total_block
7th: thread_per_block
*/
int main(int argc, char* argv[]) {

	int nodes, edges, max_edgewt, degree_frame, total_block, thread_per_block;
	double dummy_dist;
	hipError_t cudaStatus;
	nodes = atoi(argv[2]);
	edges = atoi(argv[3]);
	max_edgewt = atoi(argv[4]);
	printf("max edge weight: %d \n", max_edgewt);
	degree_frame = atoi(argv[5]);
	printf("frame degree: %d \n", degree_frame);
	dummy_dist = max_edgewt * (nodes - 1) + 1; //should be greater than (max edge_wt) * (n-1)
	total_block = atoi(argv[6]);
	thread_per_block = atoi(argv[7]);

	/*dummy_dist = 1677721401;*/
	printf("dummy dist: %f \n", dummy_dist);
	/*** Read original Graph ***/
	//int* colStartPtr_R;
	//cudaStatus = hipMallocManaged((void**)&colStartPtr_R, (nodes + 1) * sizeof(int)); //we take nodes +1 to store the start ptr of the first row 
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	/*goto Error;*/
	//}
	//int total_adjmatrix_size_R = edges * 2; //e.g.= (0 1 wt1), (1 0 wt1) both are same edge, but both will be there
	//Colwt2* cuda_adjlist_full_R;
	//cudaStatus = hipMallocManaged(&cuda_adjlist_full_R, total_adjmatrix_size_R * sizeof(Colwt2));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	/*goto Error;*/
	//}

	xEdge_cuda* allChange_cuda; //store all edges as edges to be inserted in the frame
	int totalChange = edges; //we consider number of edges = no. of edges to be inserted
	cudaStatus = hipMallocManaged(&allChange_cuda, totalChange * sizeof(xEdge_cuda));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at changeEdge");
		/*goto Error;*/
	}

	//use below for direct path
	/*string file1 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\fullGraph.txt";
	char* cstr1 = &file1[0];
	readin_graphU(&R, nodes, cstr1);*/

	//use below code if we use pass file name as argument
	//readin_graphU(&R, nodes, argv[1]);


	//use below code to pass the file name as relative path.
	//**keep the files in the same folder
	//string file1 = "./fullGraph.txt";
	//char* cstr1 = &file1[0];
	//readin_graphU4(colStartPtr_R, cuda_adjlist_full_R, cstr1, &nodes); //when local file used

	readin_graphU4(argv[1], &nodes, allChange_cuda); //when cmd line arg used

	cout << "Reading graph data successful" << endl;


	//Initializing  Rooted Tree Frame
	RT_Vertex* SSSP;
	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
		/*goto Error;*/
	}

	int src = 0; //the source from which the paths are computed

	//Time calculation
	auto startTime = high_resolution_clock::now();
	helperFunction << < 1, 1 >> > (total_block, thread_per_block, nodes, src, SSSP, dummy_dist, degree_frame, totalChange, allChange_cuda);
	hipDeviceSynchronize();
	//Time calculation
	auto stopTime = high_resolution_clock::now();
	// Time calculation
	auto duration = duration_cast<microseconds>(stopTime - startTime);
	cout << "Total time taken for GPU operations: "
		<< duration.count() << " microseconds" << endl;

	//Test code start
	cout << "SSSP" << endl;
	/*for (int i = 0; i < nodes; i++)
	{
		cout << "*******" << endl;
		cout << "node" << i << endl << "dist" << SSSP[i].Dist << endl << "parent" << SSSP[i].Parent << endl;
	}*/
	for (int i = 0; i < 5; i++)
	{
		cout << "*******" << endl;
		cout << "node" << i << endl << "dist" << SSSP[i].Dist << endl << "parent" << SSSP[i].Parent << endl;
	}
	cout << "*******success*******" << endl;

	//Test code end

	hipFree(allChange_cuda);
	hipFree(SSSP);
	return 0;
}

