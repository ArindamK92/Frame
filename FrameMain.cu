#include "hip/hip_runtime.h"
/*extern "C" {} *///it will instruct the compiler to expect C linkage for your C functions, not C++ linkage.
//#include <thrust/find.h>
//#include <thrust/device_vector.h>
//#include <thrust/count.h>
//#include <thrust/copy.h>
//#include <thrust/execution_policy.h>
//#include <thrust/device_free.h>
#include <stdio.h>
//#include "all_structures.h"
#include "all_structure_cuda.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>


#include<vector>
#include <chrono> 


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;

__global__ void initialize(int nodes, int src, RT_Vertex* SSSP, double dummy_dist, int degree_frame)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int d = degree_frame;
	int gridstride = gridDim.x * blockDim.x;

	for (int index = idx; index < nodes; index += gridstride)
	{
		if (index == src) { 
			SSSP[index].Root = src;
			SSSP[index].Dist = 0.0;
		} 
		else
		{ 
			SSSP[index].Root = src;
			int temp = index - 1;
			SSSP[index].Parent = (temp)/d;//we get the floor value as Root stores int
			/*//option1: all edge weight distinct and greater than (max edge_wt) * (n-1)
			SSSP[index].EDGwt = dummy_dist + index;
			int temp_dist = dummy_dist + index;
			while (temp > 0)
			{
				temp = temp / d;
				if (temp > 0)
				{
					temp_dist = temp_dist + dummy_dist + temp;
				}
				
			}
			SSSP[index].Dist = temp_dist;
			d_UpdatedDist[index] = temp_dist;*/

			//option 2: //taking all edges equal and greater than (max edge_wt) * (n-1)
			SSSP[index].EDGwt = dummy_dist;
			double temp_dist = dummy_dist;
			while (temp > 0)
			{
				temp = temp / d;
				if (temp > 0)
				{
					temp_dist = temp_dist + dummy_dist;
				}

			}
			SSSP[index].Dist = temp_dist;

		}
		
	}
}

//The insertDeleteEdge method might connect wrong edge depending on the sequence when the edge was connected (mainly because of the synchronization related fault)
//We avoid this error by the below method without using locking approach
//The below method tries to fit the edges using relaxation step and if it can fit, a flag is raised. 
__global__ void checkInsertedEdges(int numS, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int* change_d)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int gridstride = gridDim.x * blockDim.x;

	for (int index = idx; index < numS; index += gridstride)
	{
	
			//get the edge
			int node_1 = allChange_cuda[index].node1;
			int node_2 = allChange_cuda[index].node2;
			double edgeWeight = allChange_cuda[index].edge_wt;
			//reset it to 0
			int Edgedone = 0;



			//***Below two if logic will connect the correct edges.***
			//Check if some other edge was added--mark edge to be added //check x
			/*if (d_UpdatedDist[node1] > d_UpdatedDist[node2] + edgeWeight)*/
			if (SSSP[node_2].Dist > SSSP[node_1].Dist + edgeWeight)
			{
				Edgedone = 1;
			}

			//Check if correct edge wt was written--mark edge to be added //check x
			//this step secures the synchronization issue (if parent was updated by one thread and edgeweight was updated by different thread)
			//***this issue can be mentioned in paper separately***
			if ((SSSP[node_2].Parent == node_1) && (SSSP[node_2].EDGwt > edgeWeight))
			{
				Edgedone = 1;
			}


			if (Edgedone == 1)
			{
				//Update Parent and EdgeWt
				SSSP[node_2].Parent = node_1;
				SSSP[node_2].EDGwt = edgeWeight;
				SSSP[node_2].Dist = SSSP[node_1].Dist + edgeWeight;
				//d_UpdatedDist[node1] = d_UpdatedDist[SSSP[node1].Parent] + SSSP[node1].EDGwt;
				change_d[0] = 1; //every time node dist is updated, the flag becomes 1
			}
	}
}

void edge_update(int* totalChange, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int total_block, int thread_per_block);

/*
1st arg: original graph file name
2nd arg: no. of nodes
3rd arg: no. of edges
4th arg: max edge weight
5th arg: degree of frame
6th: total_block
7th: thread_per_block
*/
int main(int argc, char* argv[]) {

	int nodes, edges, max_edgewt, degree_frame, total_block, thread_per_block;
	double dummy_dist;
	hipError_t cudaStatus;
	nodes = atoi(argv[2]);
	edges = atoi(argv[3]); 
	max_edgewt = atoi(argv[4]);
	printf("max edge weight: %d \n", max_edgewt);
	degree_frame = atoi(argv[5]);
	printf("frame degree: %d \n", degree_frame);
	dummy_dist = max_edgewt * (nodes - 1) + 1; //should be greater than (max edge_wt) * (n-1)
	total_block = atoi(argv[6]);
	thread_per_block = atoi(argv[7]);

	/*dummy_dist = 1677721401;*/
	printf("dummy dist: %f \n", dummy_dist);
	/*** Read original Graph ***/
	//int* colStartPtr_R;
	//cudaStatus = hipMallocManaged((void**)&colStartPtr_R, (nodes + 1) * sizeof(int)); //we take nodes +1 to store the start ptr of the first row 
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	/*goto Error;*/
	//}
	//int total_adjmatrix_size_R = edges * 2; //e.g.= (0 1 wt1), (1 0 wt1) both are same edge, but both will be there
	//Colwt2* cuda_adjlist_full_R;
	//cudaStatus = hipMallocManaged(&cuda_adjlist_full_R, total_adjmatrix_size_R * sizeof(Colwt2));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	/*goto Error;*/
	//}
	   	
	xEdge_cuda* allChange_cuda; //store all edges as edges to be inserted in the frame
	int totalChange = edges; //we consider number of edges = no. of edges to be inserted
	cudaStatus = hipMallocManaged(&allChange_cuda, totalChange * sizeof(xEdge_cuda));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at changeEdge");
		/*goto Error;*/
	}

	//use below for direct path
	/*string file1 = "C:\\Users\\khand\\Desktop\\PhD\\CUDA test\\Test\\test 1\\fullGraph.txt";
	char* cstr1 = &file1[0];
	readin_graphU(&R, nodes, cstr1);*/

	//use below code if we use pass file name as argument
	//readin_graphU(&R, nodes, argv[1]);


	//use below code to pass the file name as relative path.
	//**keep the files in the same folder
	//string file1 = "./fullGraph.txt";
	//char* cstr1 = &file1[0];
	//readin_graphU4(colStartPtr_R, cuda_adjlist_full_R, cstr1, &nodes); //when local file used

	readin_graphU4(argv[1], &nodes, allChange_cuda); //when cmd line arg used

	cout << "Reading graph data successful" << endl;
	

	//Initializing  Rooted Tree Frame
	RT_Vertex* SSSP;
	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
		/*goto Error;*/
	}

	int src = 0; //the source from which the paths are computed
		
	//Time calculation
	auto startTime = high_resolution_clock::now();
	//initialize << <(nodes / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (nodes, src, SSSP, dummy_dist, degree_frame); //kernet call
	initialize << <total_block, thread_per_block>> > (nodes, src, SSSP, dummy_dist, degree_frame); //kernet call
	//hipDeviceSynchronize();
	//Time calculation
	auto stopTime = high_resolution_clock::now();
	// Time calculation
	auto duration = duration_cast<microseconds>(stopTime - startTime);
	cout << "Time taken by Frame initialization: "
		<< duration.count() << " microseconds" << endl;

	
	auto startTime1 = high_resolution_clock::now();
	edge_update(&totalChange, allChange_cuda, SSSP, total_block, thread_per_block);
	//Time calculation
	auto stopTime1 = high_resolution_clock::now();
	// Time calculation
	auto duration1 = duration_cast<microseconds>(stopTime1 - startTime1);
	cout << "Time taken by edge_update function: "
		<< duration1.count() << " microseconds" << endl;

	//Test code start
	cout << "SSSP" << endl;
	for (int i = 0; i < nodes; i++)
	{
		cout << "*******" << endl;
		cout << "node" << i << endl << "dist" << SSSP[i].Dist << endl << "parent" << SSSP[i].Parent << endl;
	}
	cout << "*******success*******" << endl;

	//Test code end

	hipFree(allChange_cuda);
	hipFree(SSSP);
	return 0;
}



void edge_update(int* totalChange, xEdge_cuda* allChange_cuda, RT_Vertex* SSSP, int total_block, int thread_per_block)
{
	/*double inf = std::numeric_limits<double>::infinity();*/

	int numS = *totalChange;

	//Go over the inserted edges to see if they need to be changed. Correct edges are connected in this stage
	int* change_d = new int[1];
	int* change = new int[1];
	change[0] = 1;
	hipMalloc((void**)&change_d, 1 * sizeof(int));
	int itr = 0;
	while (change[0] == 1)
	{
		change[0] = 0;
		hipMemcpy(change_d, change, 1 * sizeof(int), hipMemcpyHostToDevice);
		checkInsertedEdges << < total_block, thread_per_block >> > (numS, allChange_cuda, SSSP, change_d);
		hipDeviceSynchronize();
		hipMemcpy(change, change_d, 1 * sizeof(int), hipMemcpyDeviceToHost);
		/*cout << "change"<< change[0]<<endl;*/
		itr++;
	}
	//Update the distances
	 //Initialize with current distance for each node
	/*updateDistance << < (numS / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (*X_size, SSSP, d_UpdatedDist, inf);
	hipDeviceSynchronize();*/

	printf("No. of iteration: %d \n", itr);
	hipFree(change_d);
	return;
}
